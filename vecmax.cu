#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cfloat>
#include <iostream>
using namespace std;
// The number of threads per blocks in the kernel
const int threads_per_block = 256;

float *out_CPU=0;
float *in_GPU=0, *out_GPU=0;

// Forward function declarations
float GPU_vector_max(float *A, int N);
float *get_random_vector(int N);
float *get_increasing_vector(int N);

void die(const char *message);
void checkError();

// Main program
int main(int argc, char **argv) {

    // Parse vector length and kernel options
    int N=threads_per_block;
    int vecN,NvecN;
    int vp=0;
    float *tmpv;

    // Seed the random generator (for repeatable results)
    int seed;
    if(argc == 3) {
        vecN = atoi(argv[1]); // user-specified value
        seed = atoi(argv[2]);
    } else {
        die("USAGE: ./vecmax <vector_length> <rand_seed>");
    }

    NvecN= vecN/N; //Modified
    tmpv= new float[N];
    memset(tmpv,0,sizeof(float)*N);
    srand(seed);

    //float *vec = get_random_vector(N);

    // for debug, use the arrary of 0.0, 1.0, 2.0, ... as input
    // float *vec = get_increasing_vector(N);
    for (int i=0;i< NvecN;i++) {
        float *vec =get_random_vector(N); // get_increasing_vector(N);
        float result_GPU = GPU_vector_max(vec, N);
        tmpv[vp++] = result_GPU;
        if (vp > N-1) {
                float result_GPU = GPU_vector_max(tmpv, N);
                vp=0;
                memset(tmpv,0,sizeof(float)*N);
                tmpv[vp]=result_GPU;
                vp++;
        }
        hipFree(vec);
    }

    if (vecN%N) {
        float *vec =get_random_vector(vecN%N); // get_increasing_vector(N);
        float result_GPU = GPU_vector_max(vec, vecN%N);
        tmpv[vp++] = result_GPU;
        if (vp > N-1) {
                float result_GPU = GPU_vector_max(tmpv, N);
                vp=0;
                memset(tmpv,0,sizeof(float)*N);
                tmpv[vp]=result_GPU;
                vp++;
        }
    }

    if (vp > 1) {
        float result_GPU = GPU_vector_max(tmpv, N);
        tmpv[0] = result_GPU;
    }
    std:: cout << "max = "<<tmpv[0] << std::endl;
    delete tmpv;
    hipFree(in_GPU);
    hipFree(out_GPU);
    hipFree(out_CPU);
}


// A simple GPU kernel that computes the maximum value of a vector
// (each lead thread (threadIdx.x == 0) computes a single value
__global__ void vector_max_kernel(float *in, float *out, int N) {

    __shared__ float f[threads_per_block];
    // Determine the "flattened" block id and thread id
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;

    f[thread_id]=in[thread_id];
    __syncthreads();
    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block
    float max = 0.0;
    if (threadIdx.x == 0) {
        //calculate out of bounds guard
        int end = threads_per_block;
        if(thread_id + threads_per_block > N)
            end = N - thread_id;
        max = f[thread_id];

        for(int i = 1; i < end; i++) {
            if(max < f[thread_id + i])
                max = f[thread_id + i];
        }
        out[block_id] = max;
    }
}

// Returns the maximum value within a vector of length N
float GPU_vector_max(float *in_CPU, int N){

    int vector_size = N * sizeof(float);

    if(! out_CPU)
                hipHostMalloc((void **) &out_CPU, vector_size , hipHostMallocDefault);
    if (out_CPU == NULL)
                die("Error allocating CPU memory");

    if(!in_GPU) {
        if (hipMalloc((void **) &in_GPU, vector_size) != hipSuccess)
                die("Error allocating GPU memory");
    }

    if(!out_GPU) {
        if (hipMalloc((void **) &out_GPU, vector_size) != hipSuccess)
                die("Error allocating GPU memory");
    }

    hipMemcpy(in_GPU, in_CPU, vector_size, hipMemcpyHostToDevice);

    // Determine the number of thread blocks in the x- and y-dimension
    int num_blocks = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
    int max_blocks_per_dimension = 65535;
    int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blocks_per_dimension);
    int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
    dim3 grid_size(num_blocks_x, num_blocks_y, 1);

    vector_max_kernel <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);


    hipMemcpy(out_CPU, out_GPU, vector_size, hipMemcpyDeviceToHost);

    float max = out_CPU[0];

    return max;
}
// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
    if (N < 1) die("Number of elements must be greater than zero");

    float *V;
    hipHostMalloc((void **) &V, N * sizeof(float), hipHostMallocDefault);
    if (V == NULL) die("Error allocating CPU memory");

    for (int i = 0; i < N; i++) {
      V[i] = (float) rand() / (1+(float) rand());
      cout<<"V["<<i<<"] = "<<V[i]<<endl;
    }

    return V;
}

void checkError() {
    // Check for kernel errors
    hipError_t error = hipGetLastError();
    if (error) {
        char message[256];
        sprintf(message, "CUDA error: %s", hipGetErrorString(error));
        die(message);
    }
}

float *get_increasing_vector(int N) {
    if (N < 1) die("Number of elements must be greater than zero");

    float *V;
    hipHostMalloc((void **) &V, N * sizeof(float), hipHostMallocDefault);
    if (V == NULL) die("Error allocating CPU memory");

    for (int i = 0; i < N; i++) V[i] = (float) i;

    return V;
}

// Prints the specified message and quits
void die(const char *message) {
    printf("%s\n", message);
    exit(1);
}


