
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>

using namespace std;
#define n_blocks 1 
#define BLOCK_SIZE 1
//We may change this value!!
const int FILTER_WIDTH = 7;



//We may change this value!!!
int FILTER[FILTER_WIDTH*FILTER_WIDTH] = {
	1,4,7,10,7,4,1,
	4,12,26,33,26,12,4,
	7,26,55,71,55,26,7,
	10,33,71,91,71,33,10,
	7,26,55,71,55,26,7,
	4,12,26,33,26,12,4,
	1,4,7,10,7,4,1
};


// Display the first and last 10 items
// For debug only
void displayResult(const int original[], const int result[], int size) {
	cout << "Display result: ";
	cout << "(original -> result)\n";

	for (int i = 0; i < 10; i++) {
		cout << original[i] << " -> " << result[i] << "\n";
	}
	cout << ".\n.\n.\n";

	for (int i = size - 10; i < size; i++) {
		cout << original[i] << " -> " << result[i] << "\n";
	}
}

void initColorData(string file, int **data, int *sizeX, int *sizeY) {
	int x;
	int y;
	long long i = 0;
	cout << "Reading "<< file << "... \n";
	ifstream myfile(file);
	if (myfile.is_open()) {
		myfile >> x;
		myfile >> y;

		int *temp = new int[x * y * 3];
		for( i=0; i < x * y * 3; i++){
			myfile >> temp[(int)i];
		}
		myfile.close();
		*data = temp;
		*sizeX = x;
		*sizeY = y;
	}
	else {
		cout << "ERROR: File " << file << " not found!\n";
		exit(0);
	}
	cout << i << " entries imported\n";
}

void saveResult(string file, int data[], int sizeX, int sizeY) {
	long long i = 0;
	cout << "Saving data to "<< file <<"... \n";
	ofstream myfile(file, std::ofstream::out);
	if (myfile.is_open()) {
		myfile << sizeX << "\n";
		myfile << sizeY << "\n";
		for (i = 0; i < sizeX * sizeY; i++){
			myfile << data[3* i] << " " << data[3* i + 1] << " " << data[3* i+ 2]<< "\n";
		}
		myfile.close();
	}
	else {
		cout << "ERROR: Cannot save to " << file << "!\n";
		exit(0);
	}
	cout << i << " entries saved\n";
}

// TODO: implement the kneral function for 2D smoothing 

__global__ void smoothen(int *d_data, int *d_result, int *filter, int d_sizeX, int d_sizeY, int fwidth, int sum) {

      int x = blockIdx.x * blockDim.x + threadIdx.x;
      int y = blockIdx.y * blockDim.y + threadIdx.y;
      int xf,yf,hf=fwidth/2;

      __syncthreads(); // Synchronize (ensure all the data is available)

      if(y < d_sizeY && x <d_sizeX ) {
           int b=0;
           for (yf=fwidth-1;yf>=0;yf--) {
               for (xf=fwidth-1;xf>=0;xf--) {
			//What is the logic here?
                        int xt= max(0,min(x+(-xf+hf)*3,d_sizeX-1));
                        int yt= max(0,min(y-yf+hf,d_sizeY-1));
                        int w = filter[yf*fwidth+xf];
                        b += w*d_data[yt*d_sizeX+xt];
               }
           }
	   b /= sum;
           d_result[y*d_sizeX+x]=b;
        }



}


// GPU implementation
void GPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the image

	// TODO: allocate device memory and copy data onto the device
	int *d_data, *d_result, *d_filter, c, sum=0;
	int size=3*sizeX*sizeY*sizeof(int);
        hipMalloc((void **)&d_data, size);
        hipMalloc((void **)&d_result, size);
	hipMalloc((void **)&d_filter, FILTER_WIDTH*FILTER_WIDTH*sizeof(int));

        hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
        hipMemcpy(d_result, result, size, hipMemcpyHostToDevice);
	hipMemcpy(d_filter, FILTER, FILTER_WIDTH*FILTER_WIDTH*sizeof(int), hipMemcpyHostToDevice);

	// Start timer for kernel
	auto startKernel = chrono::steady_clock::now();

	// TODO: call the kernel function
	dim3 dimBlock(16,16);
	dim3 dimGrid((sizeX*3+15)/16,(sizeY+15)/16);

	for (c=0;c<FILTER_WIDTH*FILTER_WIDTH;c++) {
		sum+=FILTER[c];
	}	

	smoothen<<<dimGrid, dimBlock>>>(d_data, d_result, d_filter, sizeX*3, sizeY, FILTER_WIDTH, sum);
	// End timer for kernel and display kernel time
	hipDeviceSynchronize(); // <- DO NOT REMOVE
	auto endKernel = chrono::steady_clock::now();
	cout << "Kernel Elapsed time: " << chrono::duration <double, milli>(endKernel - startKernel).count() << "ms\n";

	// TODO: copy reuslt from device to host
	hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);
	// TODO: free device memory
        hipFree(d_data);
        hipFree(d_result);
	hipFree(d_filter);
}


// CPU implementation
void CPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the image

        int c,x,y,xf,yf,sum=0;
	int hf=FILTER_WIDTH/2;
	memset(result,0,sizeof(int)*sizeX*sizeY*3);
        for(c=0;c<FILTER_WIDTH*FILTER_WIDTH;c++) {
		sum+=FILTER[c];
	}
        for (y=0;y<sizeY;y++) {
		for (x=0;x<sizeX;x++) {
			for (yf=FILTER_WIDTH-1;yf>=0;yf--) {
        			for (xf=FILTER_WIDTH-1;xf>=0;xf--) {
        				if ((y-yf+hf)<sizeY && (x-xf+hf)<sizeX && (y-yf+hf)>=0 && (x-xf+hf)>=0) {
                                                result[y*sizeX*3+x*3]+=data[(y-yf+hf)*sizeX*3 + (x-xf+hf)*3]*
                                                        FILTER[yf*FILTER_WIDTH+xf];
                                                result[y*sizeX*3+x*3+1]+=data[(y-yf+hf)*sizeX*3 + (x-xf+hf)*3+1]*
                                                        FILTER[yf*FILTER_WIDTH+xf];
                                                result[y*sizeX*3+x*3+2]+=data[(y-yf+hf)*sizeX*3 + (x-xf+hf)*3+2]*
                                                        FILTER[yf*FILTER_WIDTH+xf];
                                        }
                                }
                        }
			result[y*sizeX*3+x*3]/=sum;
			result[y*sizeX*3+x*3+1]/=sum;
                        result[y*sizeX*3+x*3+2]/=sum;
                }
        }


	// TODO: smooth the image with filter size = FILTER_WIDTH
	//       apply zero padding for the border
}

// The image is flattened into a text file of pixel values.
int main(int argc, char *argv[]) {
	string inputFile = (argc == 1) ? "image_color.txt" : argv[1];

	int sizeX;
	int sizeY;
	int *dataForCPUTest;
	int *dataForGPUTest;	

	initColorData(inputFile, &dataForCPUTest, &sizeX, &sizeY);
	initColorData(inputFile, &dataForGPUTest, &sizeX, &sizeY);

	int size = sizeX * sizeY * 3;
	int *resultForCPUTest = new int[size];
	int *resultForGPUTest = new int[size];

	cout << "\n";

	cout << "CPU Implementation\n";

	auto startCPU = chrono::steady_clock::now();
	CPU_Test(dataForCPUTest, resultForCPUTest, sizeX, sizeY);
	auto endCPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endCPU - startCPU).count() << "ms\n";

	// displayResult(dataForCPUTest, resultForCPUTest, size);

	saveResult("color_result_CPU.txt",resultForCPUTest, sizeX, sizeY);

	cout << "\n";
	cout << "GPU Implementation\n";

	auto startGPU = chrono::steady_clock::now();
	GPU_Test(dataForGPUTest, resultForGPUTest, sizeX, sizeY);
	auto endGPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endGPU - startGPU).count() << "ms\n";

	// displayResult(dataForGPUTest, resultForGPUTest, size);
	saveResult("color_result_GPU.txt",resultForGPUTest, sizeX, sizeY);

        delete resultForCPUTest;
        delete resultForGPUTest;
        delete dataForCPUTest;
        delete dataForGPUTest;
	
	return 0;
}
